#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>

#include <hip/hip_runtime.h>

//#include <hip/hip_runtime_api.h>
//#include <helper_functions.h>

#include <hiprand.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <thrust/transform.h>
#include <thrust/sequence.h>
#include <thrust/complex.h>

#include <stdio.h>

#include <iostream>
#include <functional>

#define CELLSIZE 6
#define VECDIM 3 

static __device__ __host__ inline hipfftComplex operator+(hipfftComplex, hipfftComplex);
static __device__ __host__ inline hipfftComplex operator*(hipfftComplex, float);
static __device__ __host__ inline hipfftComplex operator*(hipfftComplex, hipfftComplex);

static __global__ void Convolution_calc(hipfftComplex*, hipfftComplex* );

int main() {
    grid_dimX = 4096;
    grid_dimY = 128;
    grid_dimZ = 3;

    const long long int ext_dimX = 2 * grid_dimX;
    const long long int ext_dimY = 2 * grid_dimY;
    const long long int ext_dimZ = 2 * grid_dimZ;
    const long long int FULLSIZE = ext_dimX * ext_dimY * ext_dimZ * 6;

    int error_value;

    //input[b * idist + ((z * inembed[1] + y) * inembed[2] + x) * istride]

    //output[b * odist + ((z * onembed[1] + y) * onembed[2] + x) * ostride]

    thrust::host_vector<hipfftComplex> host_grid(FULLSIZE);
    for (int z = 0; z < ext_dimZ; z++) {
        for (int y = 0; y < ext_dimY; y++) {
            for (int x = 0; x < ext_dimX; x++) {
                for (int b = 0; b < 6; b++) {
                    host_grid[z * ext_dimX * ext_dimY*6 + y * ext_dimX*6 + x*6 + b].x = (b + 1.f);
                }
            }
        }
    }


    long long int n[3] = { ext_dimZ, ext_dimY, ext_dimX };
    long long int inembed[3] = { 1, ext_dimY, ext_dimX };
    long long int idist = 1;
    long long int istride = 6;

    hipSetDevice(0);

    float timerValueGPU, timerValueCPU;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);


    thrust::device_vector<hipfftComplex> input_device_grid = host_grid;
    thrust::device_vector<hipfftComplex> output_device_grid(FULLSIZE);

    hipfftHandle plan_adv;
    size_t workSize;
    hipfftCreate(&plan_adv);
    hipfftXtMakePlanMany(plan_adv, 3, n, inembed, istride, idist, HIP_C_32F, inembed, istride, idist, HIP_C_32F, 6, &workSize, HIP_C_32F);
    printf("Temporary buffer size %li bytes\n", workSize);

    hipEventRecord(start, 0);

    error_value = hipfftExecC2C(plan_adv, (hipfftComplex*)thrust::raw_pointer_cast(input_device_grid.data()), (hipfftComplex*)thrust::raw_pointer_cast(input_device_grid.data()), HIPFFT_FORWARD);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&timerValueGPU, start, stop);
    printf("\n GPU calculation time %f msec\n", timerValueGPU);

    return 0;
}
////////////////////////////////////////////////////////////////////////////////
// Complex operations
////////////////////////////////////////////////////////////////////////////////

// Complex addition
static __device__ __host__ inline hipfftComplex operator+(hipfftComplex a, hipfftComplex b) {
    hipfftComplex c;
    c.x = a.x + b.x;
    c.y = a.y + b.y;
    return c;
}

// Complex scale
static __device__ __host__ inline hipfftComplex operator*(hipfftComplex a, float s) {
    hipfftComplex c;
    c.x = s * a.x;
    c.y = s * a.y;
    return c;
}

// Complex multiplication
static __device__ __host__ inline hipfftComplex operator*(hipfftComplex a, hipfftComplex b) {
    hipfftComplex c;
    c.x = a.x * b.x - a.y * b.y;
    c.y = a.x * b.y + a.y * b.x;
    return c;
}
